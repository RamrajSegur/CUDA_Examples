/* Using the CUDA framework for the array addition in the GPU*/
/* Allocate the memory in device, declare the addition function in device, pass the array from host to device,
Call the device function, Copy the results to the host, Free the device memory*/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  10

__global__ void add(int *a, int *b, int *c){
	int tid=blockIdx.x; // Used to get the index of the thread kernel
	if(tid<N)
		c[tid]=a[tid]+b[tid];
}

int main()
{
	int a[N],b[N],c[N];
	int *devA,*devB,*devC;
	
	// Enter the values in the arrays
	for(int i=0;i<N;i++)
	{
		a[i]=-i;
		b[i]=2*i;
	}
	
	//Allocate the memory in the device
	
	hipMalloc(&devA,N*sizeof(int));
	hipMalloc(&devB,N*sizeof(int));
	hipMalloc(&devC,N*sizeof(int));
		
	//Copy the values to the device memory
	
	hipMemcpy(devA , a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devB , b, N*sizeof(int), hipMemcpyHostToDevice);
	
	//Call the device function
	
	add<<<N,1>>>(devA,devB,devC);
	
	//Copy the results into the host memory location
	
	hipMemcpy(c,devC,N*sizeof(int),hipMemcpyDeviceToHost);
	
	for(int i=0;i<N;i++)
	{
		printf("A + B = C: %d + %d = %d \n" ,a[i],b[i],c[i]);
	}
	
	//Free the memory alloted in the device
	
	hipFree(devA); 
	hipFree(devB); 
	hipFree(devC);
	
	return 0;
}