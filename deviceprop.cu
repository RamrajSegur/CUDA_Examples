/* It is essential for the programmer to choose the correct GPU device from a list of GPU devices 
available to perform the calculations very efficiently, 
For Example: To run the double precision applications devices with compute capacity of more than 1.3 are desired.
The following program utilizes the cudaDeviceProp structure to get the info on the same*/

// Note: There are lot of properties available with the cudaDeviceProp struct


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

int main()
{
	hipDeviceProp_t prop;
	
	int count;
	hipGetDeviceCount(&count); //Gives the number of devices
	std::cout<<count<<std::endl;
	for(int i=0;i<count;i++)
	{
		hipGetDeviceProperties(&prop,i);
		printf( "Name: %s\n", prop.name ); // Gives the name of the device
		printf("Computation Capability: %d.%d\n", prop.major,prop.minor); // Givees the compute capability of the device
	}
	
}	