/*In order to choose the correct device for computation it is essential to compare the computation
capability of the devices available  and choose the best. It is a boring task to print out all the device properties and compare
hence CUDA C provides the methods for the same.
cudaChooseDevice is used to select the best of the devices after comparing with the desired properties-- here 1.3 computation capability
cudaSetDevice is used to set the best device. */


#include <hip/hip_runtime.h>
#include<stdio.h>

int main()
{
	hipDeviceProp_t prop;
	int dev;
	int count;
	hipGetDeviceCount(&count);
	printf("Number of devices available: %d\n",count); // Gives the number of devices available to choose from
	hipGetDevice(&dev);
	printf("ID of the current device: %d\n", dev); //ID of the current device under check
	memset(&prop,0,sizeof(hipDeviceProp_t));
	prop.major=1;
	prop.minor=3;
	hipChooseDevice(&dev,&prop); 
	printf("ID of CUDA Device Closest to 1.3 Computation Capability: %d\n", dev); // ID of the best selected device
	hipGetDeviceProperties(&prop,dev);
	printf("Computation Capability of the selected device: %d.%d\n", prop.major,prop.minor); // Gives the compute capability of the device
	hipSetDevice(dev);
}